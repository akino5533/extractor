//This file contains wrappers for C/CUDA functions to be called from Fortran.
#include <stdio.h>

#ifndef NO_GPU

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//Protect the C function names from name mangling:
#ifdef __cplusplus
extern "C"{
#endif
 void cudagetdevicecount(int* count, int* err_code);
 void cudasetdevice(int device, int* err_code);
 void cudagetdeviceproperties(int device, size_t *totalGlobalMem_, size_t *sharedMemPerBlock_,
       int *regsPerBlock_, int *warpSize_, int *maxThreadsPerBlock_, int *maxThreadsDim_, int *maxGridSize_,
       int *clockRate_, size_t *totalConstMem_, int *major_, int *minor_, int *deviceOverlap_,
       int *multiProcessorCount_, int *concurrentKernels_, int *ECCEnabled_, int *asyncEngineCount_,
       int *memoryClockRate_, int *memoryBusWidth_, int *maxThreadsPerMultiProcessor_, int* err_code);
 void cudadevicesynchronize(int *err_code);
#ifdef __cplusplus
}
#endif

#endif

#ifdef __cplusplus
extern "C"{
#endif
 int string_len(const char * str);
 char * ptr_offset(char * byte_ptr, size_t byte_offset);
 size_t c_ptr_value(void * c_ptr);
 void c_ptr_set(size_t cpval, void ** cptr);
 void print_c_ptr(void * c_ptr);
#ifdef __cplusplus
}
#endif
//-------------------------------------------------------------------
#ifndef NO_GPU
//C Wrappers (called from Fortran to invoke CUDA run-time functions):
void cudagetdevicecount(int* count, int* err_code){
 hipError_t err; const char* err_msg;
 *err_code=0;
 err=hipGetDeviceCount(count); if(err!=hipSuccess){
  err_msg=hipGetErrorString(err);
  printf("#ERROR(cudagetdevicecount): %s \n",err_msg);
  *err_code=1;
 };
 return;
}

void cudasetdevice(int device, int* err_code){
 hipError_t err; const char* err_msg;
 *err_code=0;
 err=hipSetDevice(device); if(err!=hipSuccess){
  err_msg=hipGetErrorString(err);
  printf("#ERROR(cudasetdevice): %s \n",err_msg);
  *err_code=1;
 };
 return;
}

void cudagetdeviceproperties(int device, size_t *totalGlobalMem_, size_t *sharedMemPerBlock_,
               int *regsPerBlock_, int *warpSize_, int *maxThreadsPerBlock_, int *maxThreadsDim_, int *maxGridSize_,
               int *clockRate_, size_t *totalConstMem_, int *major_, int *minor_, int *deviceOverlap_,
               int *multiProcessorCount_, int *concurrentKernels_, int *ECCEnabled_, int *asyncEngineCount_,
               int *memoryClockRate_, int *memoryBusWidth_, int *maxThreadsPerMultiProcessor_, int* err_code){
 hipError_t err; const char* err_msg; hipDeviceProp_t prop;
 *err_code=0;
 err=hipGetDeviceProperties(&prop,device);
 if(err!=hipSuccess){
  err_msg=hipGetErrorString(err);
  printf("#ERROR(cudagetdeviceproperties): %s \n",err_msg);
  *err_code=1;
 }else{
  *totalGlobalMem_=prop.totalGlobalMem;
  *sharedMemPerBlock_=prop.sharedMemPerBlock;
  *regsPerBlock_=prop.regsPerBlock;
  *warpSize_=prop.warpSize;
  *maxThreadsPerBlock_=prop.maxThreadsPerBlock;
  maxThreadsDim_[0]=prop.maxThreadsDim[0]; maxThreadsDim_[1]=prop.maxThreadsDim[1]; maxThreadsDim_[2]=prop.maxThreadsDim[2];
  maxGridSize_[0]=prop.maxGridSize[0]; maxGridSize_[1]=prop.maxGridSize[1]; maxGridSize_[2]=prop.maxGridSize[2];
  *clockRate_=prop.clockRate;
  *totalConstMem_=prop.totalConstMem;
  *major_=prop.major; *minor_=prop.minor;
  *deviceOverlap_=prop.deviceOverlap;
  *multiProcessorCount_=prop.multiProcessorCount;
  *concurrentKernels_=prop.concurrentKernels;
  *ECCEnabled_=prop.ECCEnabled;
  *asyncEngineCount_=prop.asyncEngineCount;
  *memoryClockRate_=prop.memoryClockRate;
  *memoryBusWidth_=prop.memoryBusWidth;
  *maxThreadsPerMultiProcessor_=prop.maxThreadsPerMultiProcessor;
 };
 return;
}

void cudadevicesynchronize(int *err_code)
{
 *err_code=0;
 hipError_t err=hipDeviceSynchronize(); if(err != hipSuccess){*err_code=1;}
 return;
}
#endif
//----------------------------------------------------------------------------
//Auxiliary functions:
int string_len(const char* str){ //get the length of a C string
 const int max_string_len=2147483647;
 int i;
 for(i=0;i<max_string_len;i++){if(str[i]==0) break;};
 return i;
}

char* ptr_offset(char *byte_ptr, size_t byte_offset){ //offsets a C pointer by a number of bytes
 char *addr=&byte_ptr[byte_offset];
 return addr;
}

size_t c_ptr_value(void * c_ptr){ //returns a C pointer as an integer
 return (size_t)c_ptr;
}

void c_ptr_set(size_t cpval, void ** cptr){ //sets a C pointer to a specific integer address
 *cptr=(void*)cpval;
 return;
}

void print_c_ptr(void * c_ptr){ //prints a C-pointer
 printf("%p",c_ptr);
 return;
}
